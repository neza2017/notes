#include <hip/hip_runtime.h>

#include <iostream>
#include <vector>

extern "C" __global__ void switch_test(long *src_ptr, long *dst_ptr, int num_rows, int* output_rows)
{
    int idx;
    long val;
    int pos_start = blockIdx.x * blockDim.x + threadIdx.x;
    if(pos_start >= num_rows) return;
    val = src_ptr[pos_start];
    switch(val){
        case 7016889694419943424L:
        case 3688448094816436224L:
        case 3761631588761206784L:
        case 7089228763434582016L:
        case 7161567832449220608L:{
			idx = atomicAdd(output_rows,1);
			dst_ptr[idx] = val;
			break;
		}
        default: break;
	}
	return;
}

int main()
{
    int64_t *src_ptr,*dst_ptr;
	int32_t num_rows = 10;
	auto cuda_err = hipMallocManaged(&src_ptr,sizeof(int64_t)*num_rows);
	if(cuda_err) return -6;
	std::cout << "alloc src_ptr success" << std::endl;

	cuda_err = hipMallocManaged(&dst_ptr,sizeof(int64_t)*num_rows);
	if(cuda_err) return -7;
	std::cout << "alloc dst_ptr success" << std::endl;

	int32_t* output_rows;
	cuda_err = hipMallocManaged(&output_rows,sizeof(int32_t));

	// for(int i=0;i<num_rows;++i) src_ptr[i] = i;
	src_ptr[0] = 3688729569793146880L;
	src_ptr[1] = 3617516400685350912L;
	src_ptr[2] = 3688729569793146880L;
	src_ptr[3] = 3761631588761206784L;
	src_ptr[4] = 3688448094816436224L;
	src_ptr[5] = 3689292519746568192L;
	src_ptr[6] = 3618642300592193536L;
	src_ptr[7] = 3618360825615482880L;
	src_ptr[8] = 3688729569793146880L;
	src_ptr[9] = 3617516400685350912L;
	*output_rows = 0;

    switch_test<<<2,16>>>(src_ptr,dst_ptr,num_rows,output_rows);
    hipDeviceSynchronize();
    cuda_err = hipGetLastError();
    if(cuda_err){
        std::cout << "hipDeviceSynchronize failed, error code = " << cuda_err << std::endl;
	}
	std::cout << "hipDeviceSynchronize success" << std::endl;

    for(int i=0; i<*output_rows; ++i){
		std::cout << i << ":" << dst_ptr[i] << std::endl;
	}
    return 0;
}
